#include "hip/hip_runtime.h"
/**
 * @file testhuff.cu
 * @author Cody Rivera
 * @brief
 * @version 0.0
 * @date 2022-04-13
 * (created) 2022-04-13

 * @copyright (C) 2022 by Washington State University, Argonne National Laboratory
 * See LICENSE in top-level directory
 *
 */

#include <iostream>
#include <fstream>
#include <limits>
#include <string>
#include <cstdlib>
#include <sys/stat.h>
#include "common.hh"
#include "utils.hh"
#include "huffman_coarse.cuh"
#include "huffman_parbook.cuh"

using UInt64 = unsigned long long;

size_t file_size(std::string filename) {
    struct stat stat_buf;
    int rc = stat(filename.c_str(), &stat_buf);
    return (size_t) (rc == 0 ? stat_buf.st_size : 0);
}

template <typename T, typename H, typename M = uint32_t>
void do_compress(string in_file, string out_file, int n_syms, size_t len) {
    using namespace cusz;
    int num_syms = std::min(n_syms, (int) std::numeric_limits<T>::max());

    Capsule<T> in_symbols(len, "Input symbols");
    Capsule<H> codebook(num_syms, "Codebook");
    Capsule<uint8_t> revbook(HuffmanCoarse<T, H>::get_revbook_nbyte(num_syms), "Reverse codebook");

    in_symbols.template alloc<cusz::LOC::HOST_DEVICE>();
    codebook.template alloc<cusz::LOC::DEVICE>();
    revbook.template alloc<cusz::LOC::HOST_DEVICE>();

    in_symbols.template from_file<cusz::LOC::HOST>(in_file)
        .host2device();

    HuffmanCoarse<T, H> codec;
    int sublen, pardeg;
    AutoconfigHelper::autotune(len, sublen, pardeg);
    codec.allocate_workspace(len, num_syms, pardeg);

    uint8_t* d_out;
    size_t out_len;
    codec.encode(
        in_symbols.template get<cusz::LOC::DEVICE>(),
        len,
        num_syms,
        sublen,
        pardeg,
        d_out,
        out_len
    );

    Capsule<uint8_t> out_codewords(out_len, "Output codewords");
    out_codewords.template set<cusz::LOC::DEVICE>(d_out)
        .template alloc<cusz::LOC::HOST>()
        .device2host();

    out_codewords.template to_file<cusz::LOC::HOST>(out_file + ".enc");

    in_symbols.template free<cusz::LOC::HOST_DEVICE>();
    codebook.template free<cusz::LOC::DEVICE>();
    revbook.template free<cusz::LOC::HOST_DEVICE>();
    out_codewords.template free<cusz::LOC::HOST_DEVICE>();
}

template <typename T, typename H, typename M = uint32_t>
void do_decompress(string in_file, string out_file, size_t len) {
    using namespace cusz;
    size_t in_len = file_size(in_file) / sizeof(uint8_t);

    Capsule<uint8_t> in_codewords(in_len, "Input codewords");
    Capsule<T> out_symbols(len, "Output symbols");

    in_codewords.template alloc<cusz::LOC::HOST_DEVICE>()
        .template from_file<cusz::LOC::HOST>(in_file)
        .host2device();

    out_symbols.template alloc<cusz::LOC::HOST_DEVICE>();

    HuffmanCoarse<T, H> codec;
    codec.decode(
        in_codewords.template get<cusz::LOC::DEVICE>(),
        out_symbols.template get<cusz::LOC::DEVICE>()
    );

    out_symbols.device2host();
    out_symbols.template to_file<cusz::LOC::HOST>(out_file + ".dec");

    in_codewords.template free<cusz::LOC::HOST_DEVICE>();
    out_symbols.template free<cusz::LOC::HOST_DEVICE>();
}


int main(int argc, char** argv) { 
    using namespace std;

    if (argc == 1) {
        cerr << "Usage: " << argv[0] << "(-z | -x) infile -l length [-o outfile] [--symtype (-s) u8|u16] [--cwtype (-c) u32|u64] [--nsyms (-n) num]" << endl;
        exit(0);
    }

    string in_file = "";
    string out_file = "";
    string sym_type = "u16", cw_type = "u32";
    string mode = "";
    int n_syms = 1024;
    size_t len = 0;

    int i = 1;
    while (i < argc) {
        if (argv[i][0] == '-') {
            auto long_opt = string(argv[i]);
            switch (argv[i][1]) {
                // ----------------------------------------------------------------
                case '-':
                    // string list
                    if (long_opt == "--symtype") goto sym_type;
                    if (long_opt == "--cwtype") goto cw_type;
                    if (long_opt == "--nsyms") goto n_syms;
                // ----------------------------------------------------------------
                case 'o':
                    if (i + 1 < argc) {
                        out_file = string(argv[i + 1]);
                        ++i;
                    }
                    break;
                case 's':
                sym_type:
                    if (i + 1 < argc) {
                        sym_type = string(argv[i + 1]);
                        ++i;
                    }
                    break;
                case 'c':
                cw_type:
                    if (i + 1 < argc) {
                        cw_type = string(argv[i + 1]);
                        ++i;
                    }
                    break;
                case 'n':
                n_syms:
                    if (i + 1 < argc) {
                        n_syms = stoi(string(argv[i + 1]));
                        ++i;
                    }
                    break;
                case 'l':
                    if (i + 1 < argc) {
                        len = (size_t) stoull(string(argv[i + 1]));
                        ++i;
                    }
                    break;
                case 'z':
                    mode = "compress";
                    break;
                case 'x':
                    mode = "decompress";
                    break;
                default:
                    cerr << "Bad option: " << argv[i] << endl;
                    exit(-1);
                    break;
            }
        }
        else {
            in_file = string(argv[i]);
        }
        ++i;
    }

    if (in_file == "") {
        cerr << "No input file, or no length specified" << endl;
        exit(-1);
    }

    if (out_file == "") {
        out_file = in_file;
    }

    if (mode == "compress") {
        if (len == 0) {
            cerr << "No length specified" << endl;
            exit(-1);
        }

        if (sym_type == "u8") {
            if (cw_type == "u32") {
                do_compress<uint8_t, uint32_t>(in_file, out_file, n_syms, len);
            } else if (cw_type == "u64") {
                do_compress<uint8_t, UInt64>(in_file, out_file, n_syms, len);
            }
        } else if (sym_type == "u16") {
            if (cw_type == "u32") {
                do_compress<uint16_t, uint32_t>(in_file, out_file, n_syms, len);
            } else if (cw_type == "u64") {
                do_compress<uint16_t, UInt64>(in_file, out_file, n_syms, len);
            }
        }
    } else {
        if (len == 0) {
            cerr << "No length specified" << endl;
            exit(-1);
        }

        if (sym_type == "u8") {
            if (cw_type == "u32") {
                do_decompress<uint8_t, uint32_t>(in_file, out_file, len);
            } else if (cw_type == "u64") {
                do_decompress<uint8_t, UInt64>(in_file, out_file, len);
            }
        } else if (sym_type == "u16") {
            if (cw_type == "u32") {
                do_decompress<uint16_t, uint32_t>(in_file, out_file, len);
            } else if (cw_type == "u64") {
                do_decompress<uint16_t, UInt64>(in_file, out_file, len);
            }
        }
    }

    return 0;
}
